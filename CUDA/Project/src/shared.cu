/* 
 * Course: High Performance Computing 2021/2022
 * 
 * Lecturer: Francesco Moscato	fmoscato@unisa.it
 *
 * Group:
 * Salvatore Grimaldi       0622701742      s.grimaldi29@studenti.unisa.it              
 * Enrico Maria Di Mauro    0622701706      e.dimauro5@studenti.unisa.it
 * Allegra Cuzzocrea        0622701707      a.cuzzocrea2@studenti.unisa.it
 * 
 * 
 * Copyright (C) 2021 - All Rights Reserved 
 *
 * This file is part of Contest-CUDA.
 *
 * Contest-CUDA is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * Contest-CUDA is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with Contest-CUDA.  If not, see <http://www.gnu.org/licenses/>.
 */

/**
  @file shared.cu
  @brief This is the file shared.cu, which contains the main function and the other functions
  @copyright Copyright (c) 2021
*/


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <time.h>
#include <math.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define MAXBLOCKSIZE 1024 //by Tesla K80 documentation

//useful macro for error handling
#define CUDA_CHECK(X)                                               \
  {                                                                 \
    hipError_t _m_cudaStat = X;                                    \
    if (hipSuccess != _m_cudaStat)                                 \
    {                                                               \
      fprintf(stderr, "\nErrorCuda: %s in file %s line %d\n",       \
              hipGetErrorString(_m_cudaStat), __FILE__, __LINE__); \
      exit(1);                                                      \
    }                                                               \
  }

/**
 * @brief This is the kernel that creates random numbers and insert them in 'arrayA'.
 * @param arrayA      pointer to the unsorted array.
 * @param n           number of array elements.
 * @param range       maximum acceptable integer.
 * @param seed        seed of random number.
 */
__global__ void gpu_initArray(int *arrayA, int n, int range, int seed)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandState_t state;
  hiprand_init(seed + i, 0, 0, &state);
  if (i >= n)
    return; //don't go beyond array limit
  arrayA[i] = hiprand(&state) % (range + 1);
}

/**
 * @brief This is the kernel that fulls 'arrayC' adding 1 to 'arrayC' positions which correspond to 'arrayA' elements.
 * @param arrayA      pointer to the unsorted array.
 * @param arrayC      pointer to the auxiliary array.
 * @param n           number of array elements.
 */
__global__ void gpu_fullC(int *arrayA, int *arrayC, int n, int lenC)
{
  extern __shared__ int C_shared[];

  int tid = threadIdx.x;
  for (int i = tid; i < lenC; i += blockDim.x)
    C_shared[i] = 0;
  __syncthreads();

  int input_idx = blockIdx.x * blockDim.x + tid;
  if (input_idx < n)
    atomicAdd(&C_shared[arrayA[input_idx]], 1);
  __syncthreads();

  for (int i = tid; i < lenC; i += blockDim.x)
    atomicAdd(&arrayC[i], C_shared[i]);
}

/**
 * @brief This is the kernel that sums every 'arrayC' element with the previous one.
 * @param arrayC      pointer to the auxiliary array.
 * @param len         number of array elements.
 */
__global__ void gpu_sumC(int *arrayC, int len)
{
  for (int i = 1; i < len; i++)
    arrayC[i] += arrayC[i - 1];
}

/**
 * @brief This is the kernel that sorts 'arrayA' using 'arrayC' and puts the result in 'sorted'.
 * @param arrayA      pointer to the unsorted array.
 * @param arrayC      pointer to the auxiliary array.
 * @param sorted      pointer to the sorted array.
 * @param n           number of array elements.
 */
__global__ void gpu_lastKernel(int *arrayA, int *arrayC, int *sorted, int n)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n)
    return;
  int num = arrayA[i];
  int app = atomicSub(&arrayC[num], 1); //'app' holds old arrayC[num] value (before atomicSub)
  sorted[app - 1] = num;
}

/**
 * @brief This is the function that creates and initializes a random array, calling the appropriate kernel, and puts it in 'array_h'.
 * @param array_h       pointer to the unsorted array.
 * @param n             number of array elements.
 * @param range         maximum acceptable integer.
 * @param blockSize     number of threads in each block.
 */
float initArray(int *array_h, int n, int range, int blockSize)
{
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  int *array_d; //device array

  CUDA_CHECK(hipMalloc((void **)&array_d, n * sizeof(int)));

  dim3 block(blockSize);
  dim3 grid((n - 1) / block.x + 1);

  hipError_t myCudaError;
  myCudaError = hipGetLastError(); //call done to reset previous CUDA errors

  //calling kernel to initialize array
  gpu_initArray<<<grid, block>>>(array_d, n, range, time(NULL));

  hipDeviceSynchronize(); //sync host and device
  myCudaError = hipGetLastError();
  if (myCudaError != hipSuccess)
    printf("ERROR IN gpu_initArray\n%s\n", hipGetErrorString(myCudaError));

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsed;
  hipEventElapsedTime(&elapsed, start, stop); //elapsed is the time in ms (milliseconds)
  hipEventDestroy(start);
  hipEventDestroy(stop);

  //copy to host array
  CUDA_CHECK(hipMemcpy(array_h, array_d, n * sizeof(int), hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(array_d));

  return elapsed;
}

/**
 * @brief This is the function that sorts 'array_h' using Counting Sort algorithm on the GPU.
 * @param array_h       pointer to the unsorted array.
 * @param n             number of array elements.
 * @param max           maximum acceptable integer.
 * @param blockSize     number of threads in each block.
 */
float countingSortDEVICE(int *array_h, int n, int max, int blockSize)
{
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  int *array_d;  //device array
  int *C_d;      //device array C
  int *sorted_d; //device array sorted
  int lenC = max + 1;

  CUDA_CHECK(hipMalloc((void **)&array_d, n * sizeof(int)));
  CUDA_CHECK(hipMemcpy(array_d, array_h, n * sizeof(int), hipMemcpyHostToDevice));

  CUDA_CHECK(hipMalloc((void **)&C_d, lenC * sizeof(int)));
  CUDA_CHECK(hipMemset(C_d, 0, lenC * sizeof(int))); //initialize to 0

  CUDA_CHECK(hipMalloc((void **)&sorted_d, n * sizeof(int)));

  dim3 block(blockSize);
  dim3 grid((n - 1) / block.x + 1);

  hipError_t myCudaError;
  myCudaError = hipGetLastError(); //call done to reset previous CUDA errors

  gpu_fullC<<<grid, block, sizeof(int) * lenC>>>(array_d, C_d, n, lenC);

  hipDeviceSynchronize(); //sync host and device
  myCudaError = hipGetLastError();
  if (myCudaError != hipSuccess)
    printf("ERROR IN gpu_fullC()\n%s\n", hipGetErrorString(myCudaError));

  gpu_sumC<<<1, 1>>>(C_d, lenC);

  hipDeviceSynchronize();
  myCudaError = hipGetLastError();
  if (myCudaError != hipSuccess)
    printf("ERROR IN gpu_sumC()\n%s\n", hipGetErrorString(myCudaError));

  gpu_lastKernel<<<grid, block>>>(array_d, C_d, sorted_d, n);

  hipDeviceSynchronize();
  myCudaError = hipGetLastError();
  if (myCudaError != hipSuccess)
    printf("ERROR IN gpu_lastKernelC()\n%s\n", hipGetErrorString(myCudaError));

  CUDA_CHECK(hipMemcpy(array_h, sorted_d, n * sizeof(int), hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(array_d));
  CUDA_CHECK(hipFree(C_d));
  CUDA_CHECK(hipFree(sorted_d));

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsed;
  hipEventElapsedTime(&elapsed, start, stop); //elapsed is the time in ms (milliseconds)
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return elapsed;
}

/**
 * @brief This is the function that sorts 'array' using Counting Sort algorithm on the CPU.
 * @param array         pointer to the unsorted array.
 * @param n             number of array elements.
 * @param max           maximum acceptable integer.
 */
void countingSortHOST(int *array, int n, int max)
{
  int *b;
  int *c;
  int i;
  int lenC = max + 1;

  b = (int *)malloc(n * sizeof(int));
  c = (int *)malloc(lenC * sizeof(int));

  for (i = 0; i < lenC; i++)
    c[i] = 0;

  for (i = 0; i < n; i++)
    c[array[i]] += 1;

  for (i = 1; i < lenC; i++)
    c[i] += c[i - 1];

  int num;
  for (i = 0; i < n; i++)
  {
    num = array[i];
    c[num] -= 1;
    b[c[num]] = num;
  }

  for (i = 0; i < n; i++)
  {
    array[i] = b[i];
  }

  free(b);
  free(c);
}

/**
 * @brief This is the function that creates a file ".csv" which contains values for 'blockSize', 'gridSize', 'elapsedInit', 'elapsedSort'.
 * @param blockSize       number of threads in each block.
 * @param elapsedInit     time to initialize the array.       
 * @param elapsedSort     time to sort the array
 * @param n               number of array elements.
 * @param range           maximum acceptable integer.
 */
void make_csv(int blockSize, float elapsedInit, float elapsedSort, int n, int range)
{
  FILE *fp;
  char root_filename[] = "shared_measure";
  char *filename = (char *)malloc(sizeof(char) * (strlen(root_filename) + 16 * sizeof(char)));
  sprintf(filename, "%s_%d_%d.csv", root_filename, n, range);
  if (access(filename, F_OK) == 0)
    fp = fopen(filename, "a");
  else
  {
    fp = fopen(filename, "w");
    fprintf(fp, "blockSize;gridSize;elapsedInit;elapsedSort\n");
  }
  fprintf(fp, "%d;%d;%f;%f\n", blockSize, ((n - 1) / blockSize + 1), elapsedInit / 1000, elapsedSort / 1000);
  fclose(fp);
  free(filename);
}

int main(int argc, char *argv[])
{
  int n;         //array length
  int range;     //range = max integer in array
  int blockSize; //threads per block
  int *array_h;  //host array
  int *array2_h; //host second array
  int i;
  float elapsedInit;
  float elapsedSort;

  if (argc != 4)
  {
    fprintf(stderr, "ERROR! YOU MUST INSERT ARRAY LENGTH, RANGE AND BLOCKSIZE\n");
    exit(EXIT_FAILURE);
  }

  n = atoi(argv[1]);
  range = atoi(argv[2]);
  blockSize = atoi(argv[3]);

  if (blockSize <= 0 || blockSize > MAXBLOCKSIZE)
  {
    fprintf(stderr, "ERROR! BLOCKSIZE NOT ACCEPTABLE\n");
    exit(EXIT_FAILURE);
  }

  //allocate memory on host
  array_h = (int *)malloc(n * sizeof(int));
  if (array_h == NULL)
  {
    fprintf(stderr, "ERROR! COULD NOT GET MEMORY FOR array_h\n");
  }

  array2_h = (int *)malloc(n * sizeof(int));
  if (array2_h == NULL)
  {
    fprintf(stderr, "ERROR! COULD NOT GET MEMORY FOR array2_h\n");
  }

  elapsedInit = initArray(array_h, n, range, blockSize);

  for (i = 0; i < n; i++)
    array2_h[i] = array_h[i];

  //after calling countingSort, array_h is finally sorted
  elapsedSort = countingSortDEVICE(array_h, n, range, blockSize);

  countingSortHOST(array2_h, n, range);

  for (i = 0; i < n; i++)
  {
    if (array_h[i] != array2_h[i])
    {
      printf("TEST ERROR");
      return (EXIT_FAILURE);
    }
  }

  //make_csv(blockSize, elapsedInit, elapsedSort, n, range);
  printf("%d;%d;%f;%f\n", blockSize, ((n - 1) / blockSize + 1), elapsedInit / 1000, elapsedSort / 1000);

  free(array_h);
  free(array2_h);
}